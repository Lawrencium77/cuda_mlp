#include "cuda_utils.h"

void checkCudaState(const std::string& file, int line) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error at " << file << ":" << line << " -> "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void checkCudaState(hipError_t err, const std::string& file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << " -> "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
