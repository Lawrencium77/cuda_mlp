#include "hip/hip_runtime.h"
// CUDA kernels for Matrix class
#include "matrix_kernels.h"

__global__ void matrix_add(float *a, float *b, float *c, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        c[index] = a[index] + b[index];
    }
}

__global__ void matrix_hadamard(float *a, float *b, float *c, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        c[index] = a[index] * b[index];
    }
}

// cols_a = rows_b
__global__ void matrix_multiply(float *a, float *b, float *c, int rows_a, int cols_a, int cols_b) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows_a && col < cols_b) {
        float sum = 0.0f;
        for (int k = 0; k < cols_a; k++) {
            sum += a[row * cols_a + k] * b[k * cols_b + col];
        }
        c[row * cols_b + col] = sum;
    }
}

// rows_a = len(labels)
__global__ void matrix_softmax(float *a, float* b, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows) {
        float row_sum = 0.0f;
        for (int col = 0; col < cols; col++) {
            float exp_value = expf(a[row * cols + col]);
            b[row * cols + col] = exp_value;
            row_sum += exp_value;
        }

        for (int col = 0; col < cols; col++) {
            b[row * cols + col] /= row_sum;
        }
    }
}

__global__ void matrix_sigmoid(float *a, float* b, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        b[index] = 1 / (1 + expf(-1 * a[index]));
    }
}

// Random numbers drawn from normal distribution
__global__ void fill_with_random(float *a, unsigned long seed, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        hiprandState_t state;
        hiprand_init(seed, index, 0, &state);

        float rand_normal = hiprand_normal(&state);
        a[index] = rand_normal;
    }
}
