#include "hip/hip_runtime.h"
// CUDA kernels for Matrix class
#include "matrix_kernels.h"

__global__ void matrix_const_add(float *a, float value, float *output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        output[index] = a[index] + value;
    }
}

__global__ void matrix_const_mul(float *a, float value, float *output, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        output[index] = a[index] * value;
    }
}

__global__ void matrix_sum(float* data, float* sum, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        float value = data[index];
        atomicAdd(sum, value);
    }
}


__global__ void matrix_add(float *a, float *b, float *c, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        c[index] = a[index] + b[index];
    }
}

__global__ void matrix_hadamard(float *a, float *b, float *c, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        c[index] = a[index] * b[index];
    }
}

__global__ void matrix_transpose(float *a, float *b, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        b[col * rows + row] = a[row * cols + col];
    }
}

// cols_a = rows_b
__global__ void matrix_multiply(float *a, float *b, float *c, int rows_a, int cols_a, int cols_b) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows_a && col < cols_b) {
        float sum = 0.0f;
        for (int k = 0; k < cols_a; k++) {
            sum += a[row * cols_a + k] * b[k * cols_b + col];
        }
        c[row * cols_b + col] = sum;
    }
}

__global__ void matrix_softmax_over_rows(float *a, float* b, int rows, int cols) {
    int row = threadIdx.y;

    if (row < rows) {
        float row_max = a[row * cols];
        for (int col = 1; col < cols; col++) {
            float val = a[row * cols + col];
            if (val > row_max) {
                row_max = val;
            }
        }

        float row_sum = 0.0f;
        for (int col = 0; col < cols; col++) {
            float exp_value = expf(a[row * cols + col] - row_max); // subtract max for stability
            b[row * cols + col] = exp_value;
            row_sum += exp_value;
        }

        for (int col = 0; col < cols; col++) {
            b[row * cols + col] /= row_sum;
        }
    }
}

__global__ void matrix_sigmoid(float *a, float* b, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        b[index] = 1 / (1 + expf(-1 * a[index]));
    }
}

__global__ void matrix_relu(float *a, float* b, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        b[index] = fmaxf(0.0f, a[index]);
    }
}

__global__ void matrix_relu_backward(float *a, float *grad_output, float *grad_input, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        grad_input[index] = a[index] > 0 ? grad_output[index] : 0.0f;
    }
}

// Random numbers drawn from uniform distribution
__global__ void fill_with_random(float *a, unsigned long seed, int rows, int cols, float min, float max) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        hiprandState_t state;
        hiprand_init(seed, index, 0, &state);

        float rand_uniform = hiprand_uniform(&state);
        a[index] = min + rand_uniform * (max - min); // Scale to range
    }
}

__global__ void ce_loss(float *preds, float *labels, float *losses, int rows, int cols, const float epsilon) {
    int row = threadIdx.y;

    if (row < rows) {
        int label = (int)labels[row];
        float pred = preds[row * cols + label];
        losses[row] = -1 * logf(pred + epsilon);
    }
}

__global__ void softmax_bwd(float* labels, float* softmax_outputs, float* softmax_grads, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        int label_idx = (int)labels[row];

        // https://shivammehta25.github.io/posts/deriving-categorical-cross-entropy-and-softmax/#derivation-of-softmax
        if (col == label_idx) {
            softmax_grads[idx] = softmax_outputs[idx] - 1.0f;
        } else {
            softmax_grads[idx] = softmax_outputs[idx];
        }
    }
}
