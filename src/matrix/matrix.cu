#include "hip/hip_runtime.h"
#include "matrix.h"
#include "matrix_kernels.h"
#include <iostream>

Matrix::Matrix() : rows(0), cols(0), numel(0), data(nullptr) {}

Matrix::Matrix(int rows, int cols) : rows(rows), cols(cols), numel(rows * cols) {
    hipMalloc(&data, numel * sizeof(float));
}

Matrix::~Matrix() {
    hipFree(data);
}

void Matrix::setData(const float* host_data) {
    hipMemcpy(data, host_data, numel * sizeof(float), hipMemcpyHostToDevice);
}

void Matrix::getData(float* host_data) const {
    hipMemcpy(host_data, data, numel * sizeof(float), hipMemcpyDeviceToHost);
}

Matrix::Matrix(const Matrix& other) : rows(other.rows), cols(other.cols), numel(other.numel) {
    hipMalloc(&data, numel * sizeof(float));
    hipMemcpy(data, other.data, numel * sizeof(float), hipMemcpyDeviceToDevice);
}

Matrix& Matrix::operator=(const Matrix& other) {
    if (this != &other) {
        hipFree(data);

        rows = other.rows;
        cols = other.cols;
        numel = other.numel;

        hipMalloc(&data, numel * sizeof(float));
        hipMemcpy(data, other.data, numel * sizeof(float), hipMemcpyDeviceToDevice);
    }
    return *this;
}

float matabsmax(const Matrix& mat){
    float* d_max;
    hipMalloc(&d_max, sizeof(float));
    hipMemset(d_max, 0, sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat.cols + blockSize.x - 1) / blockSize.x,
        (mat.rows + blockSize.y - 1) / blockSize.y
    );

    matrix_max_abs<<<gridSize, blockSize>>>(mat.data, d_max, mat.rows, mat.cols);
    hipDeviceSynchronize();

    float h_sum = 0.0f;
    hipMemcpy(&h_sum, d_max, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_max);
    return h_sum;
}

float matsum(const Matrix& mat){
    float* d_sum;
    hipMalloc(&d_sum, sizeof(float));
    hipMemset(d_sum, 0, sizeof(float));

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat.cols + blockSize.x - 1) / blockSize.x,
        (mat.rows + blockSize.y - 1) / blockSize.y
    );

    matrix_sum<<<gridSize, blockSize>>>(mat.data, d_sum, mat.rows, mat.cols);
    hipDeviceSynchronize();

    float h_sum = 0.0f;
    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_sum);
    return h_sum;
}

Matrix transpose(const Matrix& mat) {
    Matrix result(mat.cols, mat.rows);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat.cols + blockSize.x - 1) / blockSize.x,
        (mat.rows + blockSize.y - 1) / blockSize.y
    );

    matrix_transpose<<<gridSize, blockSize>>>(mat.data, result.data, mat.rows, mat.cols);
    hipDeviceSynchronize();
    return result;
}

Matrix softmax(const Matrix& mat) {
    Matrix result(mat.rows, mat.cols);

    dim3 blockSize(1, 1024);
    dim3 gridSize(1, (mat.rows + 1024 - 1) / 1024);

    matrix_softmax_over_rows<<<gridSize, blockSize>>>(mat.data, result.data, mat.rows, mat.cols);
    hipDeviceSynchronize();
    return result;
};

Matrix sigmoid(const Matrix& mat) {
    Matrix result(mat.rows, mat.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat.cols + blockSize.x - 1) / blockSize.x,
        (mat.rows + blockSize.y - 1) / blockSize.y
    );

    matrix_sigmoid<<<gridSize, blockSize>>>(mat.data, result.data, mat.rows, mat.cols);
    hipDeviceSynchronize();
    return result;
};

Matrix relu(const Matrix& mat) {
    Matrix result(mat.rows, mat.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat.cols + blockSize.x - 1) / blockSize.x,
        (mat.rows + blockSize.y - 1) / blockSize.y
    );

    matrix_relu<<<gridSize, blockSize>>>(mat.data, result.data, mat.rows, mat.cols);
    hipDeviceSynchronize();
    return result;
}

Matrix operator+(const Matrix& mat, const float value) {
    Matrix result(mat.rows, mat.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat.cols - 1) / blockSize.x + 1,
        (mat.rows - 1) / blockSize.y + 1
    );

    matrix_const_add<<<gridSize, blockSize>>>(mat.data, value, result.data, mat.rows, mat.cols);
    hipDeviceSynchronize();
    return result;
}

Matrix operator*(const Matrix& mat, const float value) {
    Matrix result(mat.rows, mat.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat.cols - 1) / blockSize.x + 1,
        (mat.rows - 1) / blockSize.y + 1
    );

    matrix_const_mul<<<gridSize, blockSize>>>(mat.data, value, result.data, mat.rows, mat.cols);
    hipDeviceSynchronize();
    return result;
}

Matrix operator-(const float value, const Matrix& mat) {
    Matrix negative_matrix = mat * -1.0f;
    return negative_matrix + value;
}

Matrix operator/(const Matrix& mat, const float value) {
    float inv_value = 1 / value;
    return mat * inv_value;
}

Matrix operator+(const Matrix& mat1, const Matrix& mat2) {
    if (mat1.rows != mat2.rows || mat1.cols != mat2.cols){
        std::cerr << "Matrix dimensions must match for addition!" << std::endl;
        exit(1);
    }
    Matrix result(mat1.rows, mat1.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat1.cols - 1) / blockSize.x + 1,
        (mat1.rows - 1) / blockSize.y + 1
    );

    matrix_add<<<gridSize, blockSize>>>(mat1.data, mat2.data, result.data, mat1.rows, mat1.cols);
    hipDeviceSynchronize();
    return result;
}

Matrix operator*(const Matrix& mat1, const Matrix& mat2) {
    if (mat1.rows != mat2.rows || mat1.cols != mat2.cols){
        std::cerr << "Matrix dimensions must match for Hadamard product!" << std::endl;
        exit(1);
    }
    Matrix result(mat1.rows, mat1.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat1.cols - 1) / blockSize.x + 1,
        (mat1.rows - 1) / blockSize.y + 1
    );

    matrix_hadamard<<<gridSize, blockSize>>>(mat1.data, mat2.data, result.data, mat1.rows, mat1.cols);
    hipDeviceSynchronize();
    return result;
}

Matrix matmul(const Matrix& mat1, const Matrix& mat2) {
    if (mat1.cols != mat2.rows){
        std::cerr << "Trying to multiply two matrices with non-matchiing inner dim" << std::endl;
        exit(1);
    }

    Matrix result(mat1.rows, mat2.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat2.cols - 1) / blockSize.x + 1,
        (mat1.rows - 1) / blockSize.y + 1
    );

    matrix_multiply<<<gridSize, blockSize>>>(mat1.data, mat2.data, result.data, mat1.rows, mat1.cols, mat2.cols);
    hipDeviceSynchronize();
    return result;
};

Matrix relu_backward(const Matrix& mat1, const Matrix& grad_output) {
    Matrix grad_input(mat1.rows, mat1.cols);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (mat1.cols + blockSize.x - 1) / blockSize.x,
        (mat1.rows + blockSize.y - 1) / blockSize.y
    );

    matrix_relu_backward<<<gridSize, blockSize>>>(mat1.data, grad_output.data, grad_input.data, mat1.rows, mat1.cols);
    hipDeviceSynchronize();
    return grad_input;
}

void Matrix::random(const unsigned long seed, const float min, const float max) {
    dim3 blockSize(16, 16);
    dim3 gridSize(
        (cols + blockSize.x - 1) / blockSize.x,
        (rows + blockSize.y - 1) / blockSize.y
    );

    fill_with_random<<<gridSize, blockSize>>>(data, seed, rows, cols, min, max);
    hipDeviceSynchronize();
};

Matrix get_ce_loss(const Matrix& mat1, const Matrix& labels) {
    if (mat1.rows != labels.rows) {
        std::cerr << "Non-matching number of rows for input and labels" << std::endl;
        exit(1);
    }

    Matrix losses = Matrix(mat1.rows, 1);

    dim3 blockSize(1, 1024);
    dim3 gridSize(1, 1);

    ce_loss<<<gridSize, blockSize>>>(mat1.data, labels.data, losses.data, mat1.rows, mat1.cols);
    hipDeviceSynchronize();
    return losses;
};

//  labels => (bsz, 1) => represents the index of the correct output
//  softmax_output => (bsz, num_classes)
Matrix ce_softmax_bwd(const Matrix& labels, const Matrix& softmax_output) {
    int bsz = softmax_output.rows;
    int num_classes = softmax_output.cols;

    if (labels.rows != bsz) {
        std::cerr << "Non-matching number of rows for input and labels" << std::endl;
        exit(1);
    }

    Matrix softmax_grads = Matrix(bsz, num_classes);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        (num_classes + blockSize.x - 1) / blockSize.x,
        (bsz + blockSize.y - 1) / blockSize.y
    );

    softmax_bwd<<<gridSize, blockSize>>>(labels.data, softmax_output.data, softmax_grads.data, bsz, num_classes);
    hipDeviceSynchronize();
    return softmax_grads;
}
