#include "hip/hip_runtime.h"
#include "matrix.h"
#include "matrix_kernels.h"
#include <iostream>

// Allocator Setup
std::unique_ptr<AllocatorBase> Matrix::allocator;

static std::unique_ptr<AllocatorBase> createAllocator() {
  const char *env = std::getenv("ALLOCATOR_TYPE");
  if (env && std::string(env) == "cuda") {
    std::cerr << "[INFO] Using CudaAsyncAllocator\n";
    return std::make_unique<CudaAsyncAllocator>();
  } else {
    std::cerr << "[INFO] Using MemoryAllocator (default)\n";
    return std::make_unique<MemoryAllocator>();
  }
}

__attribute__((constructor)) static void initAllocator() {
  Matrix::allocator = createAllocator();
}

// Rest of functionality
Matrix::Matrix()
    : rows(0), cols(0), numel(0), host_data(nullptr), device_data(nullptr) {}

Matrix::Matrix(int rows, int cols)
    : rows(rows), cols(cols), numel(rows * cols) {
  host_data = new float[numel];
  device_data =
      static_cast<float *>(allocator->allocate(numel * sizeof(float)));
}

Matrix::~Matrix() {
  delete[] host_data;
  allocator->free(device_data);
}

void Matrix::toDevice() {
  hipMemcpy(device_data, host_data, numel * sizeof(float),
             hipMemcpyHostToDevice);
}

void Matrix::toHost() {
  hipMemcpy(host_data, device_data, numel * sizeof(float),
             hipMemcpyDeviceToHost);
}

void Matrix::setHostData(float *data) {
  delete[] host_data;
  host_data = data;
}

Matrix::Matrix(Matrix &&other)
    : rows(other.rows), cols(other.cols), numel(other.numel),
      host_data(other.host_data), device_data(other.device_data) {
  other.rows = 0;
  other.cols = 0;
  other.numel = 0;
  other.host_data = nullptr;
  other.device_data = nullptr;
}

Matrix &Matrix::operator=(Matrix &&other) {
  if (this != &other) {
    delete[] host_data;
    allocator->free(device_data);

    rows = other.rows;
    cols = other.cols;
    numel = other.numel;
    host_data = other.host_data;
    device_data = other.device_data;

    other.rows = 0;
    other.cols = 0;
    other.numel = 0;
    other.host_data = nullptr;
    other.device_data = nullptr;
  }
  return *this;
}

Matrix &Matrix::operator=(const Matrix &other) {
  if (this != &other) {
    // Deallocate and reallocate resources since we can't assume numel ==
    // other.numel
    delete[] host_data;
    allocator->free(device_data);

    rows = other.rows;
    cols = other.cols;
    numel = other.numel;

    host_data = new float[numel];
    std::copy(other.host_data, other.host_data + numel, host_data);

    device_data =
        static_cast<float *>(allocator->allocate(numel * sizeof(float)));
    hipMemcpy(device_data, other.device_data, numel * sizeof(float),
               hipMemcpyDeviceToDevice);
  }
  return *this;
}

void Matrix::printData(std::string message) {
  toHost();
  if (message != "") {
    std::cout << message << ": \n";
  }
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      std::cout << host_data[i * cols + j] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

float matsum(const Matrix &mat) {
  float *d_sum;
  d_sum = static_cast<float *>(Matrix::allocator->allocate(sizeof(float)));
  hipError_t memset_err = hipMemsetAsync(d_sum, 0, sizeof(float));
  CHECK_CUDA_STATE_WITH_ERR(memset_err);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat.cols + blockSize.x - 1) / blockSize.x,
                (mat.rows + blockSize.y - 1) / blockSize.y);

  matrix_sum<<<gridSize, blockSize>>>(mat.device_data, d_sum, mat.rows,
                                      mat.cols);
  hipDeviceSynchronize();
  CHECK_CUDA_STATE();

  float h_sum = 0.0f;
  hipError_t memcpy_err =
      hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);
  CHECK_CUDA_STATE_WITH_ERR(memcpy_err);

  Matrix::allocator->free(d_sum);
  hipDeviceSynchronize();
  return h_sum;
}

Matrix transpose(const Matrix &mat) {
  Matrix result(mat.cols, mat.rows);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat.cols + blockSize.x - 1) / blockSize.x,
                (mat.rows + blockSize.y - 1) / blockSize.y);

  matrix_transpose<<<gridSize, blockSize>>>(mat.device_data, result.device_data,
                                            mat.rows, mat.cols);
  CHECK_CUDA_STATE();
  return result;
}

Matrix softmax(const Matrix &mat) {
  Matrix result(mat.rows, mat.cols);

  dim3 blockSize(1, 1024);
  dim3 gridSize(1, (mat.rows + 1024 - 1) / 1024);

  matrix_softmax_over_rows<<<gridSize, blockSize>>>(
      mat.device_data, result.device_data, mat.rows, mat.cols);
  CHECK_CUDA_STATE();
  return result;
};

Matrix sigmoid(const Matrix &mat) {
  Matrix result(mat.rows, mat.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat.cols + blockSize.x - 1) / blockSize.x,
                (mat.rows + blockSize.y - 1) / blockSize.y);

  matrix_sigmoid<<<gridSize, blockSize>>>(mat.device_data, result.device_data,
                                          mat.rows, mat.cols);
  CHECK_CUDA_STATE();
  return result;
};

Matrix relu(const Matrix &mat) {
  Matrix result(mat.rows, mat.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat.cols + blockSize.x - 1) / blockSize.x,
                (mat.rows + blockSize.y - 1) / blockSize.y);

  matrix_relu<<<gridSize, blockSize>>>(mat.device_data, result.device_data,
                                       mat.rows, mat.cols);
  CHECK_CUDA_STATE();
  return result;
}

Matrix operator+(const Matrix &mat, const float value) {
  Matrix result(mat.rows, mat.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat.cols - 1) / blockSize.x + 1,
                (mat.rows - 1) / blockSize.y + 1);

  matrix_const_add<<<gridSize, blockSize>>>(
      mat.device_data, value, result.device_data, mat.rows, mat.cols);
  CHECK_CUDA_STATE();
  return result;
}

Matrix operator*(const Matrix &mat, const float value) {
  Matrix result(mat.rows, mat.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat.cols - 1) / blockSize.x + 1,
                (mat.rows - 1) / blockSize.y + 1);

  matrix_const_mul<<<gridSize, blockSize>>>(
      mat.device_data, value, result.device_data, mat.rows, mat.cols);
  CHECK_CUDA_STATE();
  return result;
}

Matrix operator-(const float value, const Matrix &mat) {
  Matrix negative_matrix = mat * -1.0f;
  return negative_matrix + value;
}

Matrix operator/(const Matrix &mat, const float value) {
  float inv_value = 1 / value;
  return mat * inv_value;
}

Matrix operator+(const Matrix &mat1, const Matrix &mat2) {
  if (mat1.rows != mat2.rows || mat1.cols != mat2.cols) {
    throw std::runtime_error("Matrix dimensions must match for addition");
  }
  Matrix result(mat1.rows, mat1.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat1.cols - 1) / blockSize.x + 1,
                (mat1.rows - 1) / blockSize.y + 1);

  matrix_add<<<gridSize, blockSize>>>(mat1.device_data, mat2.device_data,
                                      result.device_data, mat1.rows, mat1.cols);
  CHECK_CUDA_STATE();
  return result;
}

Matrix operator*(const Matrix &mat1, const Matrix &mat2) {
  if (mat1.rows != mat2.rows || mat1.cols != mat2.cols) {
    throw std::runtime_error(
        "Matrix dimensions must match for Hadamard product");
  }
  Matrix result(mat1.rows, mat1.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat1.cols - 1) / blockSize.x + 1,
                (mat1.rows - 1) / blockSize.y + 1);

  matrix_hadamard<<<gridSize, blockSize>>>(mat1.device_data, mat2.device_data,
                                           result.device_data, mat1.rows,
                                           mat1.cols);
  CHECK_CUDA_STATE();
  return result;
}

Matrix matmul(const Matrix &mat1, const Matrix &mat2) {
  if (mat1.cols != mat2.rows) {
    throw std::runtime_error(
        "Trying to multiply two matrices with non-matchiing inner dim");
  }

  Matrix result(mat1.rows, mat2.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat2.cols - 1) / blockSize.x + 1,
                (mat1.rows - 1) / blockSize.y + 1);

  matrix_multiply<<<gridSize, blockSize>>>(mat1.device_data, mat2.device_data,
                                           result.device_data, mat1.rows,
                                           mat1.cols, mat2.cols);
  CHECK_CUDA_STATE();
  return result;
};

Matrix relu_backward(const Matrix &mat1, const Matrix &grad_output) {
  Matrix grad_input(mat1.rows, mat1.cols);

  dim3 blockSize(16, 16);
  dim3 gridSize((mat1.cols + blockSize.x - 1) / blockSize.x,
                (mat1.rows + blockSize.y - 1) / blockSize.y);

  matrix_relu_backward<<<gridSize, blockSize>>>(
      mat1.device_data, grad_output.device_data, grad_input.device_data,
      mat1.rows, mat1.cols);
  CHECK_CUDA_STATE();
  return grad_input;
}

void Matrix::random(const unsigned long seed, const float min,
                    const float max) {
  dim3 blockSize(16, 16);
  dim3 gridSize((cols + blockSize.x - 1) / blockSize.x,
                (rows + blockSize.y - 1) / blockSize.y);

  fill_with_random<<<gridSize, blockSize>>>(device_data, seed, rows, cols, min,
                                            max);
  CHECK_CUDA_STATE();
};

Matrix get_ce_loss(const Matrix &mat1, const Matrix &labels) {
  if (mat1.rows != labels.rows) {
    throw std::runtime_error(
        "Non-matching number of rows for input and labels");
  }

  Matrix losses = Matrix(mat1.rows, 1);

  dim3 blockSize(1, 1024);
  dim3 gridSize(1, 1);

  ce_loss<<<gridSize, blockSize>>>(mat1.device_data, labels.device_data,
                                   losses.device_data, mat1.rows, mat1.cols);
  CHECK_CUDA_STATE();
  return losses;
};

//  labels => (bsz, 1) => represents the index of the correct output
//  softmax_output => (bsz, num_classes)
Matrix ce_softmax_bwd(const Matrix &labels, const Matrix &softmax_output) {
  int bsz = softmax_output.rows;
  int num_classes = softmax_output.cols;

  if (labels.rows != bsz) {
    throw std::runtime_error(
        "Non-matching number of rows for input and labels");
  }

  Matrix softmax_grads = Matrix(bsz, num_classes);

  dim3 blockSize(16, 16);
  dim3 gridSize((num_classes + blockSize.x - 1) / blockSize.x,
                (bsz + blockSize.y - 1) / blockSize.y);

  softmax_bwd<<<gridSize, blockSize>>>(
      labels.device_data, softmax_output.device_data, softmax_grads.device_data,
      bsz, num_classes);
  CHECK_CUDA_STATE();
  return softmax_grads;
}

std::pair<Matrix, Matrix> get_ce_loss_and_accuracy(const Matrix &mat1,
                                                   const Matrix &labels) {
  if (mat1.rows != labels.rows) {
    throw std::runtime_error(
        "Non-matching number of rows for input and labels");
  }

  Matrix losses = Matrix(mat1.rows, 1);
  Matrix predictions = Matrix(mat1.rows, 1);

  dim3 blockSize(1, 1024);
  dim3 gridSize(1, 1);

  ce_loss_and_predictions<<<gridSize, blockSize>>>(
      mat1.device_data, labels.device_data, losses.device_data,
      predictions.device_data, mat1.rows, mat1.cols);
  CHECK_CUDA_STATE();
  return std::make_pair(std::move(losses), std::move(predictions));
};
