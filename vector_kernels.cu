
#include <hip/hip_runtime.h>
// CUDA kernels for Vector class

__global__ void vector_add_const(float *a, float value, float *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] + value;
    }
}

__global__ void vector_sub_const(float *a, float value, float *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] - value;
    }
}

__global__ void vector_mul_const(float *a, float value, float *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] * value;
    }
}

__global__ void vector_div_const(float *a, float value, float *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] / value;
    }
}
