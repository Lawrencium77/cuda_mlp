// Implements Vector class

#include "vector.h"
#include "vector_kernels.h"
#include <iostream>

Vector::Vector(int size) : size(size) {
    hipMalloc(&data, size * sizeof(float));
}

Vector::~Vector() {
    hipFree(data);
}

void Vector::setData(const float* host_data) {
    hipMemcpy(data, host_data, size * sizeof(float), hipMemcpyHostToDevice);
}

void Vector::getData(float* host_data) {
    hipMemcpy(host_data, data, size * sizeof(float), hipMemcpyDeviceToHost);
}

Vector Vector::operator+(const float value) {
    Vector result(size);
    vector_add_const<<<1, size>>>(data, value, result.data, size);
    hipDeviceSynchronize();
    return result;
}
